#include "hip/hip_runtime.h"
#include <chrono>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>

#include "utils/gpu/vec3.cuh"
#include "utils/gpu/color.cuh"
#include "utils/gpu/ray.cuh"
#include "utils/gpu/sphere.cuh"
#include "utils/gpu/hit_data.cuh"

using std::cin;
using std::cout;
using namespace std::chrono;


#define BLOCK_ID (blockIdx.x * gridDim.y * gridDim.z + blockIdx.y * gridDim.z + blockIdx.z)
#define THREAD_TOTAL (blockDim.x * blockDim.y * blockDim.z) 
#define THREAD_ID (threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z)

__device__ 
vec3 sample_square(int idx){
    // Initialize cuRAND state with thread-specific values
    hiprandState state;
    hiprand_init(clock64() + idx, 0, 0, &state); // Using clock64() for randomness
    
    // Generate random float in range [0,1)
    double a1 = hiprand_uniform(&state) - 0.5;
    double a2 = hiprand_uniform(&state) - 0.5;
    return vec3(a1,a2,0);

}

__host__ __device__
color ray_color(const ray& r) {
    //default sky
    vec3 unit_direction = unit_vector(r.direction());
    auto a = 0.5*(unit_direction.y() + 1.0);
    color render_color =  (1.0-a)*color(1.0, 1.0, 1.0) + a*color(0.5, 0.7, 1.0);
    double cur_distance = inf;

    Sphere spheres[] = {
        Sphere(point3(0,0,-1), 0.5, true),
        Sphere(point3(1,0,1), 0.45, color(0.9,0.9,0.05)),
        Sphere(point3(-1,0,-0.5), 0.25, color(1,0.1,0.1)),
        Sphere(point3(0.8,0.5,-1), 0.3),
        Sphere(point3(0,-100.5,-1), 100, color(0,1,0))  
    }; 

    //Spheres
    for (int sp=0; sp<sizeof(spheres)/sizeof(Sphere); sp++){
        auto hit = spheres[sp].hit_sphere(r); 
        if(hit.hit && hit.hit_distance < cur_distance){
            cur_distance = hit.hit_distance;
            render_color = hit.hit_color;
        }
    }

    return render_color;
}

__global__
void dkernel(int image_height, int image_width, vec3 pixel00_loc, vec3 camera_center, vec3 pixel_delta_u, vec3 pixel_delta_v, color* color_data){
    int i = blockIdx.x % image_width;
    int j = blockIdx.x / image_width;
    int k = threadIdx.x;
    extern __shared__ color color_val[];       ////NOTE////
    __syncthreads();
    auto offset = sample_square(k);
    auto pixel_center = pixel00_loc + ((i + offset.x()) * pixel_delta_u) + ((j + offset.y()) * pixel_delta_v);
    auto ray_direction = pixel_center - camera_center;
    ray r(camera_center, ray_direction);
    
    color_val[k] = ray_color(r);
    __syncthreads();
    
    
    for(int off = (blockDim.x+1)/2; off >= 1; off = (off+1)/2){
        if(k < off){
            color_val[k] += color_val[k + off];
            color_val[k + off] = color();
        }
        __syncthreads();
        if(off == 1) break;
    }
    
    if(k==0){
        color_data[blockIdx.x] = translate_color(color_val[0]/(double)blockDim.x);
    }
    
}

int main(int argc, char* argv[]){

    if(argc != 3){
        std::cout << "Accepts only 2 arguments" << std::endl;
        std::cout << "gpu_tracing <image_width> <samples_per_pixel>" << std::endl;
        return 0;
    }
    
    auto start_main = high_resolution_clock::now();
    
    std::ofstream img_file("gpu_image.ppm");
    
    // Image
    int image_width = std::atoi(argv[1]);
    int samples_per_pixel = std::atoi(argv[2]);
    auto aspect_ratio = 16.0 / 9.0;
    int image_height = int(image_width / aspect_ratio);
    
    // Camera
    auto camera_center = point3(0, 0, 0);
    auto viewport_height = 2.0;
    auto viewport_width = viewport_height * (double(image_width)/image_height);
    auto focal_length = 1.0;
    
    // Viewport
    auto viewport_u = vec3(viewport_width, 0, 0);
    auto viewport_v = vec3(0, -viewport_height, 0);
    auto pixel_delta_u = viewport_u / image_width;
    auto pixel_delta_v = viewport_v / image_height;
    
    auto viewport_upper_left = camera_center - vec3(0, 0, focal_length) - viewport_u/2 - viewport_v/2;
    auto pixel00_loc = viewport_upper_left + 0.5 * (pixel_delta_u + pixel_delta_v);
    
    // Output
    img_file << "P3\n" << image_width << " " << image_height << "\n255\n";
    
    // Render
    auto start = high_resolution_clock::now();    

    ////////////////// CUDA //////////////////////
    int blocks_per_grid = image_height*image_width;
    int threads_per_block = samples_per_pixel;
    
    color* output_color = new color[image_height*image_width];
    color* output_color_device;
    hipMalloc(&output_color_device, sizeof(color)*image_height*image_width);
    
    dkernel<<<blocks_per_grid,threads_per_block,sizeof(color)*(samples_per_pixel)>>>(
        image_height,
        image_width,
        pixel00_loc,
        camera_center,
        pixel_delta_u,
        pixel_delta_v,
        output_color_device
    );
    hipDeviceSynchronize();
    
    hipMemcpy(output_color,output_color_device,sizeof(color)*image_height*image_width,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    ////////////////////////////////////////
    
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    std::cout << " Computation Time Taken : " << duration.count() << std::endl;
    
    for(int i=0; i<image_height*image_width; i++){
        img_file << output_color[i] << '\n';
    }

    delete[] output_color;
    hipFree(output_color_device);

    auto stop_main = high_resolution_clock::now();
    auto duration_main = duration_cast<microseconds>(stop_main - start_main);
    std::cout << " Total Time Taken : " << duration_main.count() << std::endl;

    std::clog << "\rDone.                 \n";

    return 0;
}